#include "hip/hip_runtime.h"
#include "stl_io.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include <fstream>
#include <math.h>
namespace stl
{
  using namespace std;

  void CudaSetDevice(int dev)
  {
    hipSetDevice(dev);
  }

  __global__ void makeAllNewThreads_kernel(long start,double pow_E,long nPoints, double *dev_pointArray, double *dev_newPoint, unsigned long phantom_pointArray_size, double *dev_charges);

  __global__ void makeAllNewThreads_kernel(long start,double pow_E,long nPoints, double *dev_pointArray, double *dev_newPoint, unsigned long phantom_pointArray_size, double *dev_charges)
  {
    long i = start+threadIdx.x + blockIdx.x*blockDim.x;
    if (i < nPoints)
    {
      //local var
      double x=dev_pointArray[SSD_1*i+0];
      double y=dev_pointArray[SSD_1*i+1];
      double z=dev_pointArray[SSD_1*i+2];
      double l=dev_pointArray[SSD_1*i+3];
      double nlT0;
      double nlT1;
      double nlT2;
      double Ex,Ey,Ez;
      double E2;
      double X,Y,Z,Q;
      double Rx,Ry,Rz;
      double r2,r;
      unsigned long k;
      Ex=0;
      Ey=0;
      Ez=0;
      for(k=0;k<phantom_pointArray_size;k++){
        X=dev_charges[k*SSD_3+0];
        Y=dev_charges[k*SSD_3+1];
        Z=dev_charges[k*SSD_3+2];
        Q=dev_charges[k*SSD_3+3];
        Rx=x-X;
        Ry=y-Y;
        Rz=z-Z;
        r2=Rx*Rx+Ry*Ry+Rz*Rz;
        r=pow(r2,pow_E);
        Ex+=(Q*Rx/r);
        Ey+=(Q*Ry/r);
        Ez+=(Q*Rz/r);	      
      }
      E2=Ex*Ex+Ey*Ey+Ez*Ez;
      E2=sqrt(E2);
      Ex=Ex/E2*l;
      Ey=Ey/E2*l;
      Ez=Ez/E2*l;
      nlT0=x+Ex;
      nlT1=y+Ey;
      nlT2=z+Ez;
      dev_newPoint[SSD_1*i+0] = nlT0;
      dev_newPoint[SSD_1*i+1] = nlT1;
      dev_newPoint[SSD_1*i+2] = nlT2;
    }
  }  
  void Stl_io::MMPoi(const char* fileName,Stl_io& phantom){
    
    MMLE_arg arg(fileName);
    long FirststLayerOri;
    FirststLayerOri=(arg.nCudaIter>0)?1:-1;
    arg.nCudaIter=abs(arg.nCudaIter);
    //--------------------------internal stl information-------------------------
    nPoints=pointArray.size()/(nLayers+1);
    //nLayers
    //--------------------------Tmp Variables-------------------------------------
    math_our::Point e1,e2,e3;
    double tmpDouble;
    math_our::Point tmpPoint;
    long offset=nLayers*nPoints;
    //----------------------------------------------------------------------------
    typedef std::vector<Group> GroupArray;
    typedef std::list<Triangle> TriangleArray;
    //--------------------------normal list-----------------------------------
    math_our::Point *nList;
    long *nNList;
    get_normals_av(&nList,&nNList);
    //--------------------------length list-----------------------------------
    double *lList;
    lList=get_lengths(fileName);
//     return;
    //--------------------------profile-----------------------------------------
    vector<double> profile;
    get_profile(profile,fileName);
    //-------------------------charges----------------------------------------
    double *charges;
    ulong nCharges;
    phantom_charge charge_pair;
    charge_pair=phantom.get_charges(arg.pow_q,FirststLayerOri,fileName);
    charges=charge_pair.charge;
    nCharges=charge_pair.nCharge;
    {
      long negChC=0;
      for(unsigned long i=0;i<nCharges;i++){
        if(charges[SSD_3*i+3]>0){
        }else{
          negChC++;
        }
      }
      if(negChC)printf("negative charge  %ld |%lu %lu\n",negChC,nCharges,(ulong)(phantom.pointArray.size()));
    }
    //--------------------------statistic----------------------------------------
    unsigned int start_time =  clock();
    //--------------------------meshing------------------------------------------
    /* unique data:
    * nList[] 	- array of normal
    * lList[] 	- array of length
    * arg_n 	- number of point in each tread
    * arg.m	- mesh maker parameter
    * phantom.pointArray.size() number of cHarged points
    * pointArray[] - array of coordinates
    * shared data
    * charges[] array of x,y,z,q
    */
    vector<math_our::Point> layerBuffer;
    double *newPoint1D;
    newPoint1D=(double*)malloc(sizeof(double)*SSD_1*nPoints);
    double *pointsCuda;
    pointsCuda=(double*)malloc(sizeof(double)*SSD_2*nPoints);
    arg.nCudaIter=profile.size();
    for(long ii=0;ii<arg.nCudaIter;ii++){
      start_time =  clock();
      long arg_n=1;
//     //--------------------------internal stl information-------------------------
      nPoints=pointArray.size()/(nLayers+1);
      pointArray.resize((nLayers+arg_n+1)*nPoints);
      offset=nLayers*nPoints;
    //--------------------------first layer--------------------------------------
      bool flag=false;
      if(!is_3Dmesh){
        if(arg.firstLayer){
          for(long i=0;i<nPoints;i++) {
            tmpDouble=nList[i].module();
            tmpPoint=nList[i]*lList[i]*profile[ii]/tmpDouble*((double)FirststLayerOri);
            pointArray[offset+nPoints+i]=pointArray[offset+i]+tmpPoint;
          }
          nLayers++;
          offset+=nPoints;
        }else{
          flag=true;
          layerBuffer.reserve(nPoints);
          for(long i=0;i<nPoints;i++) {
            tmpDouble=nList[i].module();
            tmpPoint=nList[i]*lList[i]*profile[ii]/tmpDouble*0.05*((double)FirststLayerOri);
            layerBuffer.push_back(pointArray[offset+i]);
//             pointArray[offset+i]+=tmpPoint;
          }
          arg_n++;
        }
      }
      else{
        arg_n++;
      }
      //-------------------------other layers--------------------------------------
      
      if(arg_n>1){
        for(long i=0;i<nPoints;i++){
          pointsCuda[SSD_2*i+0]=pointArray[offset+i].x;
          pointsCuda[SSD_2*i+1]=pointArray[offset+i].y;
          pointsCuda[SSD_2*i+2]=pointArray[offset+i].z;
          pointsCuda[SSD_2*i+3]=lList[i]*profile[ii];
        }
      //--------------------------cuda place---------------------------------------
        
        double *dev_charges, *dev_newPoint;
        double *dev_pointArray;

        hipMalloc( (void**)&dev_charges, SSD_3*nCharges*sizeof(double) );
        hipMalloc( (void**)&dev_pointArray, nPoints*SSD_2*sizeof(double) );
        hipMalloc( (void**)&dev_newPoint, SSD_1*nPoints*sizeof(double) );

        hipGetErrorString (hipMemcpy(dev_charges, charges, SSD_3*nCharges*sizeof(double), hipMemcpyHostToDevice));
        hipGetErrorString (hipMemcpy(dev_pointArray, pointsCuda, nPoints*SSD_2*sizeof(double), hipMemcpyHostToDevice));

        for(long internal_i=0;internal_i<nPoints/(long)2048+(long)1;++internal_i){
          long threads = 256;
          long blocks = 8;//(nPoints)/threads+1;
          long start = internal_i*2048;
          makeAllNewThreads_kernel<<<blocks, threads>>>(
            start,
            arg.pow_E,nPoints,dev_pointArray,
            dev_newPoint,
            nCharges,dev_charges);
        }
        hipGetErrorString (hipMemcpy(newPoint1D, dev_newPoint, SSD_1*nPoints*sizeof(double), hipMemcpyDeviceToHost));
        hipFree(dev_charges);
        hipFree(dev_pointArray);
        hipFree(dev_newPoint);
        for(long i=0;i<nPoints;i++){
          for(long j=1;j<arg_n;j++){
            pointArray[offset+j*nPoints+i].x=newPoint1D[SSD_1*i+0];
            pointArray[offset+j*nPoints+i].y=newPoint1D[SSD_1*i+1];
            pointArray[offset+j*nPoints+i].z=newPoint1D[SSD_1*i+2];
          }
        }
      }     
      nLayers+=arg_n;
      nLayers--;
      if(flag){
        for(long i=0;i<nPoints;i++) {
          pointArray[offset+i]=layerBuffer[i];
        }
        layerBuffer.clear();
      }
      //---------------------------relaxation--------------------------------------
      llrelax_cu_fast(arg.nStep,arg.elas,arg.elas2);
      //--------------------------after meshing actions----------------------------
      unsigned int end_time = clock();
      unsigned int search_time = end_time - start_time;
      fprintf(stdout,"Time (s) %ld,%06ld, Time per point %ld.%06ld (s). %4ld Layers\n",
              search_time/1000000,search_time%1000000,search_time/nPoints/1000000,search_time/nPoints%1000000,nLayers);
      fflush(stdout);
      is_3Dmesh=(true);
    }
    free(nList);
    free(nNList);
    free(lList);
    free(charges);
    free(newPoint1D);
    free(pointsCuda);
  }
}
