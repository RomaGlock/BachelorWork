#include "hip/hip_runtime.h"
#include "stl_io.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include <fstream>
#include <math.h>
#include "math_our_cu.h"
#define DXRELAX 0.001
namespace stl
{
  using namespace std;
  __global__ void makelrelax_fast(
          long start,
          long nPoints,
          long* dev_nnei,
          math_our_cu::Point_cu* dev_neigth_points,
          math_our_cu::Point_cu* dev_next_layer,
          double alpha,
          long mem_struct_size,
          long* dev_exept
        ){
    long i = start+threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= nPoints) return;
    long n=dev_nnei[i];
    long index;
    math_our_cu::Point_cu e,downP,bufP,x0,e0;
    x0=dev_neigth_points[mem_struct_size*i+2*n];
    downP=dev_neigth_points[mem_struct_size*i+2*n+1];
    bufP=dev_neigth_points[mem_struct_size*i+2*n+2];
    e=bufP-downP;
    e0=e;
    double tmpDouble;
    double N=double(n);
    double cs=e.module();
    e0/=cs;
    math_our_cu::Point_cu f(0.,0.,0.);
    double f_,f_elas,dfdt,dfdt_elas,f_elas_;
    math_our_cu::Point_cu tmpPoint;
    math_our_cu::Point_cu tmpPoint_;
    double tmpPointModule,tmpPoint_e;
    double tmpPointModule_,tmpPoint_e_;
    for(long j=0;j<n;++j){
      index=mem_struct_size*i+j;
      f+=dev_neigth_points[index];
    }
    for(long iter=0;iter<10;++iter){
      e=(x0-downP).normalize();
//       tmpPoint=bufP-x0;
      tmpPoint=downP+e*cs-x0;
      tmpPointModule=tmpPoint.module();
      tmpPoint_e=tmpPoint*e;
//       tmpPoint_=bufP-x0-e*DXRELAX*cs;      //tmpPoint_=bufP-x0-e*DXRELAX;
      tmpPoint_=downP+e*cs-x0-e*DXRELAX*cs;
      tmpPointModule_=tmpPoint_.module();
      tmpPoint_e_=tmpPoint_*e;
      f_=(f-x0*N)*e;
      f_elas=tmpPoint_e*tmpPointModule;
      f_elas_=tmpPoint_e_*tmpPointModule_;
      dfdt=-N;
      dfdt_elas=(f_elas_-f_elas)/DXRELAX/cs; //dfdt_elas=(f_elas_-f_elas)/DXRELAX
      f_elas/=cs;
      dfdt_elas/=cs;
      f_elas*=alpha;
      dfdt_elas*=alpha;
      tmpDouble=(f_+f_elas);
      tmpDouble/=(dfdt+dfdt_elas);
      if((((x0-e*tmpDouble)-downP)*e)>0.5*cs){
        x0-=e*tmpDouble;
      }else{
        x0=downP+e*cs*0.5;
      }
    }    
    dev_next_layer[i]=x0;
  }
  __global__ void refreshnei(
          long start,
          long nPoints,
          long* dev_nei,long* dev_nnei,long* dev_offset_nei,
          math_our_cu::Point_cu* dev_neigth_points,
          math_our_cu::Point_cu* dev_next_layer,
          long mem_struct_size
        ){
    long i = start+threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= nPoints) return;
    long n=dev_nnei[i];
    long offset=dev_offset_nei[i];
    for(ulong j=0;j<n;j++){
      dev_neigth_points[mem_struct_size*i+j]=dev_next_layer[dev_nei[offset+j]];
    }
    dev_neigth_points[mem_struct_size*i+2*n]=dev_next_layer[i];
  }
  __device__ math_our_cu::Point_cu find_force1(math_our_cu::Point_cu* nei,
                                              long n,
                                              math_our_cu::Point_cu x0,
                                              math_our_cu::Point_cu downP,
                                              math_our_cu::Point_cu bufP,
                                              math_our_cu::Point_cu normal0,
                                              double cs,
                                              double alpha,long i
                                             ){
    math_our_cu::Point_cu normal,currEdge,f(0.,0.,0.);
    double tmpDouble;
    normal=(x0-downP).normalize();
    for(ulong j=0;j<n;j++){
      currEdge=x0-nei[j];
      tmpDouble=currEdge.x*currEdge.x+currEdge.y*currEdge.y+currEdge.z*currEdge.z;
      f+=currEdge/tmpDouble*cs;
    }
    tmpDouble=(bufP-x0).module();
    if(tmpDouble>cs*0.01){
      f+=(bufP-x0)/tmpDouble*sqrt(fabs(1.-normal0*normal))*alpha;
      f+=(bufP-x0)/cs*alpha;
    }
    return f;
  }
  __device__ math_our_cu::Point_cu find_force(math_our_cu::Point_cu* nei,
                                              long nn,
                                              math_our_cu::Point_cu x0,
                                              math_our_cu::Point_cu downP,
                                              math_our_cu::Point_cu bufP,
                                              math_our_cu::Point_cu normal0,
                                              double cs,
                                              double alpha,long i, long out
                                             ){
    math_our_cu::Point_cu normal,n,currEdge,f(0.,0.,0.),sf(0.,0.,0.),uf(0.,0.,0.),a,b;
    double tmpDouble,D,l;
    normal=x0-downP;
    l=normal.module();
    normal/=l;
    for(ulong j=0;j<nn;j++){
      currEdge=nei[j]-x0;
//       tmpDouble=(nei[j-n]-downP).module();
      f+=currEdge;// /tmpDouble;
    }
    for(ulong j=0;j<(nn-1);j++){
      a=(nei[j+nn]-nei[j]);
      b=(nei[j+1+nn]-nei[j]);
      n=a^b;      
      tmpDouble=n.module();
      n/=tmpDouble;
      D=-(nei[j+nn]*n);
      if((downP-nei[j+nn])*n<0){
        n*=(-1.);
        D=-D;
      }
      currEdge=nei[j]-nei[j+1];
      sf+=n*(currEdge.module()*0.86-((x0*n)+D));
      //force side//
      a=(nei[j]-nei[j+1+nn]);
      b=(nei[j+1]-nei[j+1+nn]);
      n=a^b;
      tmpDouble=n.module();
      n/=tmpDouble;
      D=-(nei[j+1+nn]*n);
      if((downP-nei[j+1+nn])*n<0){
        n*=(-1.);
        D=-D;
      }
      sf+=n*(currEdge.module()*0.86-((x0*n)+D));
      //force side//
      a=(nei[j+nn]-downP);
      b=(nei[j+1+nn]-downP);
      n=a^b;
      tmpDouble=n.module();
      n/=tmpDouble;
      D=-(downP*n);
      if((bufP-downP)*n<0){
        n*=(-1.);
        D=-D;
      }
      uf+=n*(l-((x0*n)+D));
      //force up//
    }
    if(i){
      a=(nei[nn-1+nn]-nei[nn-1]);
      b=(nei[0+nn]-nei[nn-1]);
      n=a^b;
      tmpDouble=n.module();
      n/=tmpDouble;
      D=-(nei[nn-1+nn]*n);
      if((downP-nei[nn-1+nn])*n<0){
        n*=(-1.);
        D=-D;
      }
      currEdge=nei[nn-1]-nei[0];
      sf+=n*(currEdge.module()*0.86-((x0*n)+D));
      //force side//   
      a=(nei[nn-1]-nei[0+nn]);
      b=(nei[0]-nei[0+nn]);
      n=a^b;
      tmpDouble=n.module();
      n/=tmpDouble;
      D=-(nei[0+nn]*n);
      if((downP-nei[0+nn])*n<0){
        n*=(-1.);
        D=-D;
      }
      sf+=n*(currEdge.module()*0.86-((x0*n)+D));
      //force side//
      a=(nei[nn-1+nn]-downP);
      b=(nei[0+nn]-downP);
      n=a^b;
      tmpDouble=n.module();
      n/=tmpDouble;
      D=-(downP*n);
      if((bufP-downP)*n<0){
        n*=(-1.);
        D=-D;
      }
      uf+=n*(l-((x0*n)+D));      
      //force up//
    }
//     tmpDouble=(bufP-x0).module();
//     if(tmpDouble>cs*0.01){
//       f+=(bufP-x0)/tmpDouble*sqrt(fabs(1.-normal0*normal))*alpha*cs;
//       f+=(bufP-x0)*alpha;
//     }
//     return f+sf*alpha+uf*alpha;
    return f+uf*alpha+sf*alpha;
  }
  __device__ math_our_cu::Point_cu spring_relax_solve(math_our_cu::Point_cu x0,
                                                      math_our_cu::Point_cu dx,
                                                      math_our_cu::Point_cu downP,
                                                      double D,
                                                      math_our_cu::Point_cu n,
                                                      double min_cos,long i){
    double f1,f2,fmid;
    math_our_cu::Point_cu x1,x2,xmid;
    x1=x0;
    x2=x1+dx;
    f2=(x2-downP).normalize()*n-min_cos;
    if(f2>0)return dx;
    f1=(x1-downP).normalize()*n-min_cos;
    if(f1<0)return math_our_cu::Point_cu(0.,0.,0.); //no idea how to solve
    for(long iter=0;iter<10;++iter){
      xmid=(x1+x2)*0.5;
      fmid=(xmid-downP).normalize()*n-min_cos;
      if(fmid>0){
        x1=xmid;
      }else{
        x2=xmid;
      }
    }
    return (x1+x2)*0.5-x0;
  }
  __device__ math_our_cu::Point_cu spring_relax_check(math_our_cu::Point_cu x0,
                                                math_our_cu::Point_cu dx,
                                                math_our_cu::Point_cu bufP,
                                                math_our_cu::Point_cu downP,
                                                math_our_cu::Point_cu* nei,
                                                long nnei,
                                                double min_cos,long i
                                             ){
    double D;
    math_our_cu::Point_cu n;
    for(ulong j=0;j<(nnei-1);j++){
      n=(nei[j]-downP)^(nei[j+1]-downP);
      n.normalize();
      D=-(downP*n);
      if((bufP-downP)*n<0){
        n*=(-1.);
        D=-D;
      }
      
      dx=spring_relax_solve(x0,dx,downP,D,n,min_cos,i);
    }
    if(i){
      n=(nei[nnei-1]-downP)^(nei[0]-downP);
      n.normalize();
      D=-(downP*n);
      if((bufP-downP)*n<0){
        n*=(-1.);
        D=-D;
      }
      dx=spring_relax_solve(x0,dx,downP,D,n,min_cos,i);
    }
    return dx;
  }
  __global__ void spring_relax(
          long start,
          long nPoints,
          long* dev_nnei,
          math_our_cu::Point_cu* dev_neigth_points,
          math_our_cu::Point_cu* dev_next_layer,
          double alpha,
          long mem_struct_size,
          long* dev_exept
        ){
    long i = start+threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= nPoints) return;
    long out=0;
    long n=dev_nnei[i];
    long offset_curr=mem_struct_size*i;
    long offset_prev=offset_curr+n;
    math_our_cu::Point_cu x0=dev_neigth_points[offset_prev+n];
    if(!dev_exept[i]){
//       dev_next_layer[i]=x0;
      return;
    }
    math_our_cu::Point_cu downP=dev_neigth_points[offset_prev+n+1];
    math_our_cu::Point_cu bufP=dev_neigth_points[offset_prev+n+2];
    math_our_cu::Point_cu f,f_dx,f_dy;
    math_our_cu::Point_cu force_dir_x,force_dir_y;
    math_our_cu::Point_cu x_dx,x_dy;
    double cs,f_n,tmpDouble;
    double J11,J12,J21,J22;
    double J_11,J_21;
    double dx,dy;
    math_our_cu::Point_cu normal0=bufP-downP;
    cs=normal0.module();
    normal0/=cs;
    math_our_cu::Point_cu normal;
    for(long iter=0;iter<1;++iter){
      normal=(x0-downP).normalize();
      f=find_force(dev_neigth_points+offset_curr,n,x0,downP,bufP,normal0,cs,alpha,1,out);
      if( (isnan(f.x)) && (isnan(f.y)) && (isnan(f.z))){
        printf("f   %ld\n",i);
      }
      f=f-(normal*(f*normal));
      f_n=f.module();     //force module
      if(f_n<1e-7){
        dev_next_layer[i]=x0; 
        return;
      }
      force_dir_x=f/f_n;  //force direction
      force_dir_y=normal^force_dir_x;
      x_dx=x0+force_dir_x*DXRELAX*cs;
      x_dy=x0+force_dir_y*DXRELAX*cs;
      f_dx=find_force(dev_neigth_points+offset_curr,n,x_dx,downP,bufP,normal0,cs,alpha,1,out);
      if( (isnan(f_dx.x)) && (isnan(f_dx.y)) && (isnan(f_dx.z))){
        printf("fdx %ld\n",i);
      }
      f_dy=find_force(dev_neigth_points+offset_curr,n,x_dy,downP,bufP,normal0,cs,alpha,1,out);
      if( (isnan(f_dy.x)) && (isnan(f_dy.y)) && (isnan(f_dy.z))){
        printf("fdy %ld\n",i);
      }
      J11=(f_dx*force_dir_x-f_n)/(DXRELAX*cs);
      J12=(f_dy*force_dir_x-f_n)/(DXRELAX*cs);
      J21=(f_dx*force_dir_y -0.)/(DXRELAX*cs);
      J22=(f_dy*force_dir_y -0.)/(DXRELAX*cs);
      tmpDouble=J11*J22-J12*J21;
      J_11=J22/tmpDouble;
      J_21=-J21/tmpDouble;
      dx=J_11*f_n;
      dy=J_21*f_n;
      x_dx=force_dir_x*dx*(-1)-force_dir_y*dy;
      x_dx=spring_relax_check(x0,x_dx,bufP,downP,dev_neigth_points+offset_prev,n,0.3,1);
      if( (!isnan(x_dx.x)) && (!isnan(x_dx.y)) && (!isnan(x_dx.z))){
        x0+=x_dx;
      }
    }
    dev_next_layer[i]=x0;    
  }
  __global__ void spring_relax_edge(
          long start,
          long nPoints,
          long* dev_nnei,
          math_our_cu::Point_cu* dev_neigth_points,
          math_our_cu::Point_cu* dev_next_layer,
          double alpha,
          long mem_struct_size,
          long* dev_exept
        ){
    long i = start+threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= nPoints) return;
    long out=0;
    if(i==13737){
      out=1;
    }
    long n=dev_nnei[i];
    long offset_curr=mem_struct_size*i;
    long offset_prev=offset_curr+n;
    math_our_cu::Point_cu x0=dev_neigth_points[offset_prev+n];
    if(dev_exept[i]){
      return;
    }
    math_our_cu::Point_cu downP=dev_neigth_points[offset_prev+n+1];
    math_our_cu::Point_cu bufP=dev_neigth_points[offset_prev+n+2];
    math_our_cu::Point_cu f,f_dx;
    math_our_cu::Point_cu force_dir_x;
    math_our_cu::Point_cu x_dx;
    math_our_cu::Point_cu edge_n;
    double cs,f_n;
    double J11;
    double dx;
    math_our_cu::Point_cu normal0=bufP-downP;
    cs=normal0.module();
    normal0/=cs;
    edge_n=(dev_neigth_points[offset_prev]-bufP)^(dev_neigth_points[offset_prev+n-1]-bufP);
    edge_n.normalize();
//     force_dir_x=(edge_n^normal0).normalize();
    math_our_cu::Point_cu normal;
    for(long iter=0;iter<2;++iter){
      normal=(x0-downP).normalize();
      force_dir_x=(edge_n^normal).normalize();
      f=find_force(dev_neigth_points+offset_curr,n,x0,downP,bufP,normal0,cs,alpha,0,out);
      if( (isnan(f.x)) && (isnan(f.y)) && (isnan(f.z))){
        printf("f   %ld\n",i);
      }
      f_n=f*force_dir_x;
      if(fabs(f_n)<1e-10){
        dev_next_layer[i]=x0; 
        return;
      }
      x_dx=x0+force_dir_x*DXRELAX*cs;
      f_dx=find_force(dev_neigth_points+offset_curr,n,x_dx,downP,bufP,normal0,cs,alpha,0,out);
      if( (isnan(f_dx.x)) && (isnan(f_dx.y)) && (isnan(f_dx.z))){
        printf("fdx   %ld\n",i);
      }
      J11=(f_dx*force_dir_x-f_n)/(DXRELAX*cs);
      dx=-f_n/J11;
      x_dx=force_dir_x*dx;
      x_dx=spring_relax_check(x0,x_dx,bufP,downP,dev_neigth_points+offset_prev,n,0.3,0);
      if(out){
        printf("f    %lf %lf %lf\n",f.x,f.y,f.z);
        printf("f_dx %lf %lf %lf\n",f_dx.x,f_dx.y,f_dx.z);
        printf("n %lf %lf %lf\n",normal.x,normal.y,normal.z);
        printf("f_n %lf J11 %lf dx %lf x+dx %lf\n",f_n,J11,dx,(x0+x_dx-downP).module());
        printf("xdx   %lf %lf %lf\n",x_dx.x,x_dx.y,x_dx.z);
      }
      if( (!isnan(x_dx.x)) && (!isnan(x_dx.y)) && (!isnan(x_dx.z))){
        x0+=x_dx;
      }
    }
    dev_next_layer[i]=x0;
  }
  void Stl_io::llrelax_cu_fast(long nStep,double elas,double elas2){
    vector<math_our::Point> layerBuffer;
    math_our::Point tmpPoint;
//     double tmpDouble;
    long tmpLong;
    //------------------------exeption list--------------------------------------------
    ulong k=0;
    set<edge> edgeTree;
    for(GroupArray::const_iterator i=groupArray.begin();i!=groupArray.end();++i){
      for(Group::TriangleArray::const_iterator j = i->tri.begin(); j != i->tri.end(); ++j) {
        addTriInEdgeTree(k,*j,edgeTree);
        k++;
      }
    }
    //------------------------------make owner<neighbour--------------------------
    long* exept;
    exept=(long*)malloc(nPoints*sizeof(long));
    for(long i=0;i<nPoints;i++){
      exept[i]=1;
    }
    for(set<edge>::iterator i=edgeTree.begin();i!=edgeTree.end();++i){
      if(i->neig==-1){
        exept[i->a]=0;
        exept[i->b]=0;
      }
    }  
    //--------------------------neighbours------------------------------------
    vector<vector<long> > neiList;
    vector<long> pointNeiPoint;
    vector<long> nei_offset;
    vector<long> pointNNeiPoint;
    pointNNeiPoint.reserve(nPoints);
    neiList.resize(nPoints);
    nei_offset.reserve(nPoints);
    
    findPnP(neiList);
    long pNP=0;
    nei_offset.push_back(0);
    ulong max_nnei=0;
    for(long i=0;i<nPoints;++i){
      pointNNeiPoint.push_back(neiList[i].size());
      pNP+=neiList[i].size();
      max_nnei=max_nnei>neiList[i].size()?max_nnei:neiList[i].size();
      if(i>0){
        nei_offset.push_back(nei_offset[i-1]+pointNNeiPoint[i-1]);
      }
    }
    pointNeiPoint.reserve(pNP);
    for(long i=0;i<nPoints;++i){
      pointNeiPoint.insert(pointNeiPoint.end(),neiList[i].begin(),neiList[i].end());
    }
    layerBuffer.reserve(nPoints);
    layerBuffer.insert<math_our::Point*>(layerBuffer.end(),&(pointArray[nLayers*nPoints]),&(pointArray[nPoints*(nLayers+1)]));
    math_our::Point* layer_result_buffer;
    math_our::Point *neigth_points;
    layer_result_buffer=(math_our::Point*)malloc(nPoints*sizeof(math_our::Point));
    long mem_struct_size=(long)(2*max_nnei+4);
    neigth_points=(math_our::Point*)malloc(mem_struct_size*nPoints*sizeof(math_our::Point));
    ulong tmpUlong;
    for(long i=0;i<nPoints;i++){
      tmpUlong=neiList[i].size();
      for(ulong j=0;j<tmpUlong;j++){
        tmpLong=pointNeiPoint[nei_offset[i]+j];
        tmpPoint=layerBuffer[tmpLong];
        neigth_points[mem_struct_size*i+j]=tmpPoint;
        tmpPoint=pointArray[(nLayers-1)*nPoints+tmpLong];
        neigth_points[mem_struct_size*i+j+tmpUlong]=tmpPoint;
      }
      tmpPoint=layerBuffer[i];
      neigth_points[mem_struct_size*i+2*tmpUlong]=tmpPoint;
      neigth_points[mem_struct_size*i+2*tmpUlong+2]=tmpPoint;
      tmpPoint=pointArray[(nLayers-1)*nPoints+i];
      neigth_points[mem_struct_size*i+2*tmpUlong+1]=tmpPoint;
      
    }      
    //---------------------------------------------------------------------------------
    long* dev_nei;
    long* dev_nnei;
    long* dev_offset_nei;
    long* dev_exept;
    math_our_cu::Point_cu* dev_neigth_points;
    math_our_cu::Point_cu* dev_next_layer;
    hipMalloc( (void**)&dev_nei, pNP*sizeof(long) );
    hipMalloc( (void**)&dev_nnei, nPoints*sizeof(long) );
    hipMalloc( (void**)&dev_exept, nPoints*sizeof(long) );
    hipMalloc( (void**)&dev_offset_nei, nPoints*sizeof(long) );
    hipMalloc( (void**)&dev_next_layer, nPoints*sizeof(math_our::Point) );
    hipMalloc( (void**)&dev_neigth_points,mem_struct_size*nPoints*sizeof(math_our_cu::Point_cu) );
    hipMemcpy((void*)dev_nei, (void*)&(pointNeiPoint[0]), pNP*sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy((void*)dev_offset_nei, (void*)&(nei_offset[0]), nPoints*sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy((void*)dev_nnei, (void*)&(pointNNeiPoint[0]), nPoints*sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy((void*)dev_exept, (void*)&(exept[0]), nPoints*sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy((void*)dev_next_layer, (void*)&(layerBuffer[0]), nPoints*sizeof(math_our::Point), hipMemcpyHostToDevice);
    hipMemcpy((void*)dev_neigth_points, (void*)&(neigth_points[0]), mem_struct_size*nPoints*sizeof(math_our_cu::Point_cu), hipMemcpyHostToDevice);
    ulong mem_used=mem_struct_size*nPoints*sizeof(math_our_cu::Point_cu)+nPoints*sizeof(math_our::Point)+nPoints*sizeof(long);    
    printf("mem used %10lu KB\n",mem_used/1024);
    ulong start_time =  clock();
    for(long n_elas=0;n_elas<nStep;++n_elas){
      for(long internal_i=0;internal_i<nPoints/(long)2048+(long)1;++internal_i){
        long threads = 256;
        long blocks = 8;//(nPoints)/threads+1;
        long start = internal_i*2048;
        makelrelax_fast<<<blocks, threads>>>(
          start,
          nPoints,
          dev_nnei,
          dev_neigth_points,
          dev_next_layer,
          elas,
          mem_struct_size,
          dev_exept
        );
      }      
      for(long internal_i=0;internal_i<nPoints/(long)2048+(long)1;++internal_i){
        long threads = 256;
        long blocks = 8;//(nPoints)/threads+1;
        long start = internal_i*2048;
        refreshnei<<<blocks, threads>>>(
          start,
          nPoints,
          dev_nei,dev_nnei,dev_offset_nei,
          dev_neigth_points,
          dev_next_layer,
          mem_struct_size
        );
      }
      for(long internal_i=0;internal_i<nPoints/(long)2048+(long)1;++internal_i){
        long threads = 256;
        long blocks = 8;//(nPoints)/threads+1;
        long start = internal_i*2048;
        spring_relax<<<blocks, threads>>>(
          start,
          nPoints,
          dev_nnei,
          dev_neigth_points,
          dev_next_layer,
          elas2,
          mem_struct_size,
          dev_exept
        );
      }
      for(long internal_i=0;internal_i<nPoints/(long)2048+(long)1;++internal_i){
        long threads = 256;
        long blocks = 8;//(nPoints)/threads+1;
        long start = internal_i*2048;
        spring_relax_edge<<<blocks, threads>>>(
          start,
          nPoints,
          dev_nnei,
          dev_neigth_points,
          dev_next_layer,
          elas2,
          mem_struct_size,
          dev_exept
        );
      }
      for(long internal_i=0;internal_i<nPoints/(long)2048+(long)1;++internal_i){
        long threads = 256;
        long blocks = 8;//(nPoints)/threads+1;
        long start = internal_i*2048;
        refreshnei<<<blocks, threads>>>(
          start,
          nPoints,
          dev_nei,dev_nnei,dev_offset_nei,
          dev_neigth_points,
          dev_next_layer,
          mem_struct_size
        );
      }
      if(nStep>40){
        long xy=nStep/40;
        if(!(n_elas%xy)){
          printf("#");
          fflush(stdout);
        }
      }
    }
    ulong end_time = clock();
    ulong search_time = end_time - start_time;
    fprintf(stdout,"\nRelax time (s) %ld,%06ld\n",
            search_time/1000000,search_time%1000000);
    fflush(stdout);
    hipMemcpy((void*)&(pointArray[nLayers*nPoints]), (void*)dev_next_layer, nPoints*sizeof(math_our::Point), hipMemcpyDeviceToHost);
//     tmpLong=0;
//     for(long i=0;i<nPoints;i++){
//       math_our::Point p1,p2;
//       p1=pointArray[nLayers*nPoints+i]-pointArray[(nLayers-1)*nPoints+i];
//       p2=layerBuffer[i]-pointArray[(nLayers-1)*nPoints+i];
//       p1.normalize();
//       p2.normalize();
//       if(p1*p2<0.999){
//         tmpLong++;
//       }
//     }
//     printf("%ld\n",tmpLong);
    hipFree(dev_nnei);
    hipFree(dev_next_layer);
    hipFree(dev_neigth_points);
    hipFree(dev_offset_nei);
    hipFree(dev_nei);
    hipFree(dev_exept);
    free(layer_result_buffer);
    free(neigth_points);
    free(exept);
  }
}

namespace math_our_cu{
  __device__ Point_cu::Point_cu():x(0),y(0),z(0){}
  __device__ Point_cu::Point_cu(double a,double b,double c):x(a),y(b),z(c){}
  __device__ void Point_cu::set(double a,double b,double c){
    this->x=a;
    this->y=b;
    this->z=c;
  }
  
  __device__ double Point_cu::module() const{
    return norm3d(x,y,z);
  }
  __device__ Point_cu Point_cu::operator+(const Point_cu& p) const{
    return Point_cu(x+p.x, y+p.y, z+p.z);
  }
  __device__ Point_cu Point_cu::operator^(const Point_cu& p) const{
    return Point_cu(y*p.z-z*p.y, z*p.x-x*p.z, x*p.y-y*p.x);
  }
  __device__ Point_cu Point_cu::operator*(double p) const{
    return Point_cu(x*p, y*p, z*p);
  }
  __device__ Point_cu Point_cu::operator/(double p) const{
    return Point_cu(x/p, y/p, z/p);
  }
  __device__ const Point_cu & Point_cu::operator+=(const Point_cu& p){
    x += p.x;
    y += p.y;
    z += p.z;
    return *this;
  }
  __device__ const Point_cu & Point_cu::operator-=(const Point_cu& p){
    x -= p.x;
    y -= p.y;
    z -= p.z;
    return *this;
  }
  __device__ const Point_cu & Point_cu::operator/=(double p){
    x /= p;
    y /= p;
    z /= p;
    return *this;
  }
  __device__ const Point_cu & Point_cu::operator*=(double p){
    x *= p;
    y *= p;
    z *= p;
    return *this;
  }
  __device__ const Point_cu& Point_cu::normalize()
  {
    (*this) /= module();
    return *this;
  }
  __device__ Point_cu Point_cu::operator-(const Point_cu& p) const{
    return Point_cu(x-p.x, y-p.y, z-p.z);
  }
  __device__ double Point_cu::operator*(const Point_cu& p) const{
    return x*p.x+y*p.y+z*p.z;
  }
}