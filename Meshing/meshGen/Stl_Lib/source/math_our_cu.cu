#include "hip/hip_runtime.h"
//
#include "math_our_cu.h"
namespace math_our_cu{
  __device__ Point_cu::Point_cu():x(0),y(0),z(0){}
  __device__ Point_cu::Point_cu(double a,double b,double c):x(a),y(b),z(c){}
  __device__ void Point_cu::set(double a,double b,double c){
    this->x=a;
    this->y=b;
    this->z=c;
  }
  
  __device__ double Point_cu::module() const{
    return norm3d(x,y,z);
  }
  __device__ Point_cu Point_cu::operator+(const Point_cu& p) const{
    return Point_cu(x+p.x, y+p.y, z+p.z);
  }
  __device__ Point_cu Point_cu::operator^(const Point_cu& p) const{
    return Point_cu(y*p.z-z*p.y, z*p.x-x*p.z, x*p.y-y*p.x);
  }
  __device__ Point_cu Point_cu::operator*(double p) const{
    return Point_cu(x*p, y*p, z*p);
  }
  __device__ Point_cu Point_cu::operator/(double p) const{
    return Point_cu(x/p, y/p, z/p);
  }
  __device__ const Point_cu & Point_cu::operator+=(const Point_cu& p){
    x += p.x;
    y += p.y;
    z += p.z;
    return *this;
  }
  __device__ const Point_cu & Point_cu::operator-=(const Point_cu& p){
    x -= p.x;
    y -= p.y;
    z -= p.z;
    return *this;
  }
  __device__ const Point_cu & Point_cu::operator/=(double p){
    x /= p;
    y /= p;
    z /= p;
    return *this;
  }
  __device__ const Point_cu & Point_cu::operator*=(double p){
    x *= p;
    y *= p;
    z *= p;
    return *this;
  }
  __device__ const Point_cu& Point_cu::normalize()
  {
    (*this) /= module();
    return *this;
  }
  __device__ Point_cu Point_cu::operator-(const Point_cu& p) const{
    return Point_cu(x-p.x, y-p.y, z-p.z);
  }
  __device__ double Point_cu::operator*(const Point_cu& p) const{
    return x*p.x+y*p.y+z*p.z;
  }
}
